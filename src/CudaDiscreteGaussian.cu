#include "hip/hip_runtime.h"
///CudaDiscreteGaussian.cu
/**
 * \author Luis Lourenço (2010)
 * \version 3.0.0
 * \since 15/09/10
 */

#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>
#include <cutil.h>

#include "CudaDiscreteGaussian.h"


/// allocate texture variables
texture<float, 1, hipReadModeElementType> texRef;
texture<float, 1, hipReadModeElementType> gaussTexRef;



__global__ void calculateGaussianKernel(float *gaussKernel, const float sigma, int kernelWidth){

  ///pixel index of this thread
  int pixIdx = blockIdx.x * blockDim.x + threadIdx.x;
  int i = pixIdx - (kernelWidth>>1);
  float smaller;
  extern __shared__ float s_gaussKernel[];
  
  ///this kernel must allocate 'kernelWidth' threads
  s_gaussKernel[threadIdx.x] = (__fdividef(1,(sqrtf(2*M_PI*sigma))))*expf((-1)*(__fdividef((i*i),(2*sigma*sigma))));

  __syncthreads();

  smaller = s_gaussKernel[0];

  gaussKernel[pixIdx] = s_gaussKernel[threadIdx.x]/smaller;

}

__global__ void kernel_1DConvolutionH_texture(float *output, int3 size, short halfkernelsize){
///this version uses the texture memory to store the gaussian kernel and the
///image data

  float2 sum;
  int2 pos;

  extern __shared__ float s_gauss[];

  ///pixel index of this thread
  int pixIdx = blockIdx.x * blockDim.x + threadIdx.x;

  ///output pixel index
  pos.y = __fdividef(pixIdx,size.x);
  pos.x = pixIdx-(pos.y*size.x);

  if(threadIdx.x<((halfkernelsize<<1)+1)) s_gauss[threadIdx.x] = tex1Dfetch(gaussTexRef,threadIdx.x);
  
  sum.x = sum.y = 0;

  for(int k=-halfkernelsize;k<(halfkernelsize+1);k++){
    sum.x += (tex1Dfetch(texRef, pixIdx + k * (((pos.x+k)>=0)*((pos.x+k)<size.x))) * s_gauss[k+halfkernelsize]);
    sum.y += s_gauss[k+halfkernelsize];
  }

  output[pixIdx] = __fdividef(sum.x,sum.y);
}

__global__ void kernel_1DConvolutionV_texture(float *output, int3 size, short halfkernelsize){
///this version uses the texture memory to store the gaussian kernel and the
///image data

  float2 sum;
  int2 pos;

  extern __shared__ float s_gauss[];

  ///pixel index of this thread
  int pixIdx = blockIdx.x * blockDim.x + threadIdx.x;

  ///output pixel index
  pos.y = __fdividef(pixIdx,size.x);
  pos.x = pixIdx-(pos.y*size.x);

  if(threadIdx.x<((halfkernelsize<<1)+1)) s_gauss[threadIdx.x] = tex1Dfetch(gaussTexRef,threadIdx.x);
  
  sum.x = sum.y = 0;

  for(int k=-halfkernelsize;k<(halfkernelsize+1);k++){
    sum.x += (tex1Dfetch(texRef, pixIdx + (size.x*k) * (((pos.y+k)>=0)*((pos.y+k<size.y)))) * s_gauss[k+halfkernelsize]);
    sum.y += s_gauss[k+halfkernelsize];
  }

  output[pixIdx] = __fdividef(sum.x,sum.y);
}

extern "C"
float* cudaDiscreteGaussian2D(const float *d_img, int width, int height, float gaussianVariance, unsigned int maxKernelWidth){

  int3 size;
  size.x = width;
  size.y = height;
  size.z = width*height;

  int threadsPerBlock = 256;
  int blocksPerGrid = ((size.z) + threadsPerBlock -1) >> 8;
  dim3 DimBlock(threadsPerBlock,1,1);
  dim3 DimGrid(blocksPerGrid,1,1);

  int kernelSize = maxKernelWidth*sizeof(float);

  unsigned int timer = 0;
  cutCreateTimer( &timer );
  cutStartTimer( timer );  /// Start timer

  /// The Gaussian Kernel Width must be odd
  if (maxKernelWidth < 1) maxKernelWidth = 1;
  if (maxKernelWidth%2 == 0) maxKernelWidth--;
  short halfkernelsize = maxKernelWidth >> 1;

  float *cudaGaussKernel;
  hipMalloc((void**)&cudaGaussKernel,kernelSize);

  /// Calculate gaussian kernel
  calculateGaussianKernel<<<1,maxKernelWidth,kernelSize>>>(cudaGaussKernel, gaussianVariance, maxKernelWidth);

  /// Allocate output memory to image data
  float *d_output;
  hipMalloc((void**) &d_output, size.z*sizeof(float));

  /// Allocate temporary memory to image data
  float *d_tmpbuffer;
  hipMalloc((void**) &d_tmpbuffer, size.z*sizeof(float));

  /// Bind a texture to the CUDA array
  hipBindTexture (NULL, gaussTexRef, cudaGaussKernel);
  CUT_CHECK_ERROR("Texture bind failed");

  /// Host side settable texture attributes
  gaussTexRef.normalized = false;
  gaussTexRef.filterMode = hipFilterModePoint;

  /// Bind a texture to the CUDA array
  hipBindTexture (NULL, texRef, d_img);
  CUT_CHECK_ERROR("Texture bind failed");

  /// Host side settable texture attributes
  texRef.normalized = false;
  texRef.filterMode = hipFilterModePoint;

  kernel_1DConvolutionH_texture<<<DimGrid,DimBlock,kernelSize>>>(d_tmpbuffer,size,halfkernelsize);

  /// Bind temporary data texture
  hipUnbindTexture(texRef);
  hipBindTexture (NULL ,texRef, d_tmpbuffer);

  kernel_1DConvolutionV_texture<<<DimGrid,DimBlock,kernelSize>>>(d_output,size,halfkernelsize);

  /// Free allocated memory
  hipFree(d_tmpbuffer);
  hipUnbindTexture(texRef);
  hipFree(cudaGaussKernel);
  hipUnbindTexture(gaussTexRef);
  CUT_CHECK_ERROR("Memory image free failed");

  hipDeviceSynchronize();
  cutStopTimer( timer );  /// Stop timer
  printf("Gaussian time = %f ms\n",cutGetTimerValue( timer ));

  return(d_output);
}

